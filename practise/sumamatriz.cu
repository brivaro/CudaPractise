
#include <hip/hip_runtime.h>
#include <stdio.h>
#define M 8
#define N 12

__global__ void add(int *a, int *b, int *c)
{
  int tidx = threadIdx.x + blockIdx.x * blockDim.x;
  int tidy = threadIdx.y + blockIdx.y * blockDim.y;

  c[tidx + tidy * M] = a[tidx + tidy * M] + b[tidx + tidy * M];
}

int main()
{
  int a[N * M], b[N * M], c[N * M];
  int *dev_a, *dev_b, *dev_c, i, j;
  // reservar memoria en GPU
  hipMalloc((void **)&dev_a, M * N * sizeof(int));
  hipMalloc((void **)&dev_b, M * N * sizeof(int));
  hipMalloc((void **)&dev_c, M * N * sizeof(int));
  // rellenar vectores en CPU
  for (i = 0; i < M; i++)
    for (j = 0; j < N; j++)
    {
      a[i + j * M] = i + j;
      b[i + j * M] = i * i;
    }
  // enviar vectores a GPU
  hipMemcpy(dev_a, a, N * M * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b, N * M * sizeof(int), hipMemcpyHostToDevice);
  // cudaMemcpy( dev_c, c, N*sizeof(int) , cudaMemcpyHostToDevice );
  dim3 bpg(2, 3);
  dim3 tpb(4, 4);
  // llamar al Kernel
  add<<<bpg, tpb>>>(dev_a, dev_b, dev_c);
  // obtener el resultado de vuelta en la CPU
  hipMemcpy(c, dev_c, N * M * sizeof(int), hipMemcpyDeviceToHost);
  for (i = 0; i < M; i++)
    for (j = 0; j < N; j++)
      printf(" %d + %d = %d\n", a[i + j * M], b[i + j * M], c[i + j * M]);

  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);
}
