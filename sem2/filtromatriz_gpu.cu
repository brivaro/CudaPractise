
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 8
#define M 12

__global__ void filtro_gpu(float *a, float *c)
{
 // int  tidx=threadIdx.x+blockIdx.x*blockDim.x;
 // int tidy= threadIdx.y + blockIdx.y*blockDim.y;
  int  tidx=threadIdx.x;
  int tidy= threadIdx.y;
 
 
 c[tidx+tidy*(M-2)]=(a[tidx+(tidy+1)*M]+a[tidx+1+(tidy+1)*M]+a[tidx+2+(tidy+1)*M]+a[tidx+1+tidy*M]+a[tidx+1+(tidy+2)*M])/5.0;
 }


void Print_matrix(float C[], int n, int m) {
   int i, j;

   for (i = 0; i < n; i++) {
      for (j = 0; j < m; j++)
         printf("%.2e ", C[i+j*n]);
      printf("\n");
   }
}  /* Print_matrix */

void filtrocpu(float *a, float *c)
{
	int i,j;
	for (i=0;i<M-2;i++)
	  for (j=0;j<N-2;j++)
	   c[i+j*(M-2)]=(a[i+(j+1)*M]+a[i+1+(j+1)*M]+a[i+2+(j+1)*M]+a[i+1+(j)*M]+a[i+1+(j+2)*M])/5.0;
	
}
 
 int main() {
 float A[N*M], C1[(N-2)*(M-2)], C2[(N-2)*(M-2)];

 int i,j;

  for (i=0;i<M;i++)
  for (j=0;j<N;j++)
   {
     A[i+j*M]=i+j;
     
    }
printf("A \n");
Print_matrix(A,M,N);
filtrocpu(A,C1);

printf("\n C cpu \n");
Print_matrix(C1,M-2,N-2);

float *dev_A,  *dev_C;
 //reservar memoria en GPU
 hipMalloc((void **) &dev_A, M*N*sizeof(float) );
 hipMalloc((void **) &dev_C, (M-2)*(N-2)*sizeof(float) );
hipMemcpy( dev_A, A, M*N*sizeof(float) , hipMemcpyHostToDevice );
//dim3 block_p_grd(2,3);
dim3 thr_p_block(M-2,N-2);

filtro_gpu<<<1, thr_p_block>>>(dev_A,dev_C);
hipMemcpy( C2, dev_C, (M-2)*(N-2)*sizeof(float), hipMemcpyDeviceToHost );

printf("\n C gpu \n");
Print_matrix(C2,M-2,N-2);





  }
	
	
