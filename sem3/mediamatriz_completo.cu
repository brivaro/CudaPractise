
#include <hip/hip_runtime.h>
#include <stdio.h>

#define CUDA_SAFE_CALL( call ) {                                         \
 hipError_t err = call;                                                 \
 if( hipSuccess != err ) {                                              \
   fprintf(stderr,"CUDA: error occurred in cuda routine. Exiting...\n"); \
   exit(err);                                                            \
 } }

#define N 10240
#define M 1024
#define	BLOCKSIZE 32



__global__ void mediasmatriznaive(double *A,  double *sal)
{
  int i,tid=blockIdx.x; 
  double suma=0;
  if (tid <M) {
    for(i=0;i<N;i++)
       suma=suma+A[i+tid*N];
    sal[tid]=suma/double(N);
  }
 }

__global__ void mediasmatrizfast(double *A,  double *sal)
{
__shared__ double cache[BLOCKSIZE];
int tid=threadIdx.x;
int col=blockIdx.x;
int cacheindex=threadIdx.x;
double suma=0.0,temp=0.0;
while (tid<N)
{temp +=A[tid+col*N];
tid+= BLOCKSIZE;
}
cache[cacheindex]=temp; 
__syncthreads();
int i=blockDim.x/2;
while (i!=0)
{ if (cacheindex<i)
   cache[cacheindex]+=cache[cacheindex+i];
    __syncthreads();
   i=i/2;
}
if (cacheindex==0)
sal[blockIdx.x]=cache[0]/N;
 }
 
 int main() {
  int i,j;
 double *dev_A,  *dev_sal;
 
  double *A = (double *) malloc( N*M*sizeof(double) );
  double *sal1 = (double *) malloc( M*sizeof(double) );
double *sal2 = (double *) malloc( M*sizeof(double) );
 //reservar memoria en GPU
 hipMalloc((void **) &dev_A, N*M*sizeof(double) );
 hipMalloc((void **) &dev_sal, M*sizeof(double) );
 
 //rellenar matriz en CPU
  for (j=0;j<M;j++)
    for(i=0;i<N;i++)
   {
      A[i+N*j]=2.0f * ( (double) rand() / RAND_MAX ) ;
    }
//enviar vectores a GPU
hipMemcpy( dev_A, A, N*M*sizeof(double) , hipMemcpyHostToDevice );


//cudaMemcpy( dev_sal, sal, N*sizeof(double) , cudaMemcpyHostToDevice );

//llamar al Kernel1
  hipEvent_t start, stop;
  CUDA_SAFE_CALL( hipEventCreate(&start) );
  CUDA_SAFE_CALL( hipEventCreate(&stop) );
//
CUDA_SAFE_CALL( hipEventRecord(start, NULL) ); // Record the start event

 mediasmatrizfast<<<M,BLOCKSIZE>>>(dev_A,dev_sal);
 //obtener el resultado de vuelta en la CPU
 hipMemcpy( sal1, dev_sal, M*sizeof(double), hipMemcpyDeviceToHost );

CUDA_SAFE_CALL( hipEventRecord(stop, NULL) );  // Record the stop event
  CUDA_SAFE_CALL( hipEventSynchronize(stop) );   // Wait for the stop event to complete
  float msecGPU = 0.0f;
  CUDA_SAFE_CALL( hipEventElapsedTime(&msecGPU, start, stop) );
  printf("GPU time1 = %.2f msec.\n",msecGPU);
//
// kernel 2
//
CUDA_SAFE_CALL( hipEventRecord(start, NULL) ); // Record the start event

 mediasmatriznaive<<<M,1>>>(dev_A,dev_sal);
 //obtener el resultado de vuelta en la CPU
 hipMemcpy( sal2, dev_sal, M*sizeof(double), hipMemcpyDeviceToHost );

  CUDA_SAFE_CALL( hipEventRecord(stop, NULL) );  // Record the stop event
  CUDA_SAFE_CALL( hipEventSynchronize(stop) );   // Wait for the stop event to complete
  msecGPU = 0.0f;
  CUDA_SAFE_CALL( hipEventElapsedTime(&msecGPU, start, stop) );
  printf("GPU time2 = %.2f msec.\n",msecGPU);

//
  double error=fabs(sal1[0]-sal2[0]);
  double aux;
  for (i=1;i<M;i++)
      { aux=fabs(sal1[i]-sal2[i]);
        if (aux>error)
           error=aux;
      }
  printf("error %f  \n",error);
  hipFree(dev_A) ;
  hipFree(dev_sal) ;
  free(A);
  free(sal1);
 free(sal2);
  }
	
	
