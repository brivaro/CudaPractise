

#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 12
#define BLOCKSIZE 4

void Print_matrix(int C[], int n)
{
   int i, j;

   for (i = 0; i < n; i++)
   {
      for (j = 0; j < n; j++)
         printf("%d ", C[i + j * n]);
      printf("\n");
   }
} /* Print_matrix */

void comprobar_cpu(int *A, int *sal)
{
   int i, j, res = 1;
   for (j = 0; j < N - 1; j++)
      for (i = 0; i < N - 1; i++)
         if (A[i + j * N] != A[i + 1 + (j + 1) * N])
            res = 0;

   *sal = res;
}

int comprobar_cpu_vector(int *sal)
{
   int i, res = 1;
   for (i = 0; i < N-1; i++)
      if (sal[i] != sal[i + 1])
         res = 0;

   return res;
}

__global__ void kernel(int *A, int *sal)
{
   int i;
   __shared__ int cache[BLOCKSIZE];
   int tid = threadIdx.x;
   int cacheindex = threadIdx.x;
   int col = blockIdx.x;
   int res = 1;
   while (tid < N - 1 && col < N - 1)
   {
     if ((A[tid + col * N] != A[tid + 1 + (col+1) * N]))
       res=0;
     tid += BLOCKSIZE;
   }
   cache[cacheindex] = res;
   __syncthreads();
   i = blockDim.x / 2;
   while (i != 0)
   {
      if (cacheindex < i){
         if ((cache[cacheindex + i] != cache[cacheindex])){
           cache[cacheindex] = 0;
         }
       }
      __syncthreads();
      i = i / 2;
   }
   if (cacheindex == 0)
      sal[col] = cache[0];
 }

int main()
{

   int i, j;
   int *A = (int *)malloc(N * N * sizeof(int));
   int *sal = (int *)malloc(N * sizeof(int));
   int salcpu;

   // rellenar matriz de numeros en CPU
   for (j = 0; j < N; j++)
      for (i = 0; i < N; i++)
      {
         A[i + N * j] = j - i;
      }
   // A[3+N*4]=77;
   Print_matrix(A, N);
   comprobar_cpu(A, &salcpu);
   if (salcpu == 1)
      printf(" \n CPU La matriz es toeplitz \n");
   else
      printf(" \n CPU La matriz no es toeplitz \n");

   // Aqui pon el código para reservar memoria, copiar matriz, llamar kernel, traer resultados,
   //  y lo que sea necesario

   // Comienzo parte GPU
   int *dev_A, *dev_sal;

   hipMalloc((void **)&dev_A, N * N * sizeof(int));
   hipMalloc((void **)&dev_sal, N * sizeof(int));
   hipMemcpy(dev_A, A, N * N * sizeof(int), hipMemcpyHostToDevice);

   kernel<<<N,BLOCKSIZE>>>(dev_A,dev_sal);

   hipMemcpy(sal, dev_sal, N * sizeof(int), hipMemcpyDeviceToHost);

   int res = comprobar_cpu_vector(sal);
   if (res == 1)
      printf(" \n GPU La matriz es toeplitz \n");
   else
      printf(" \n GPU La matriz no es toeplitz \n");

}
