
#include <hip/hip_runtime.h>
#include <stdio.h>


#define M 5
#define N 6


__global__ void mediasmatrizgpu(double *A,  double *sal)
{
  int i; 
  int tid=threadIdx.x;
  double suma;

    suma=0;
    for(i=0;i<M;i++)
       suma=suma+A[i+tid*M];
    sal[tid]=suma/double(M);
   
 }

void Print_matrix(double C[], int m, int n) {
   int i, j;

   for (i = 0; i < m; i++) {
      for (j = 0; j < n; j++)
         printf("%.2e ", C[i+j*m]);
      printf("\n");
   }
}  /* Print_matrix */
 
 int main() {
  int i,j;
double *dev_A, *dev_sal1;
 
  double *A = (double *) malloc( N*M*sizeof(double) );
  double *sal1 = (double *) malloc( N*sizeof(double) );
hipMalloc((void **) &dev_A, M*N*sizeof(double) );
 hipMalloc((void **) &dev_sal1, N*sizeof(double) );
 
 //rellenar matriz en CPU
  for (j=0;j<N;j++)
    for(i=0;i<M;i++)
   {
      A[i+M*j]=i+j ;
    }

  Print_matrix(A,M,N);
hipMemcpy( dev_A, A, M*N*sizeof(double), hipMemcpyHostToDevice );
  mediasmatrizgpu<<<1,N>>>(dev_A,dev_sal1);
hipMemcpy( sal1, dev_sal1, N*sizeof(double), hipMemcpyDeviceToHost );
   for (j=0;j<N;j++)
    printf("media columna %d = %f  \n",j,sal1[j]);

  free(A);
  free(sal1);

  }
	
	
