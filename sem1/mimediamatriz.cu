
#include <hip/hip_runtime.h>
#include <stdio.h>


#define M 5
#define N 6


void mediasmatrizcpu(double *A,  double *sal)
{
  int i,j; 
  double suma;
  for(j=0;j<N;j++)
    {suma=0;
    for(i=0;i<M;i++)
       suma=suma+A[i+j*M];
    sal[j]=suma/double(M);
    }
 }


__global__ void mediatriz_gpu(double *A,  double *sal)
 {
   int tid = threadIdx.x;
   double suma;
   suma=0;
     for(int i=0;i<M;i++)
        suma=suma+A[i+tid*M];
   sal[tid]=suma/double(M);
   
}


void Print_matrix(double C[], int m, int n) {
   int i, j;

   for (i = 0; i < m; i++) {
      for (j = 0; j < n; j++)
         printf("%.2e ", C[i+j*m]);
      printf("\n");
   }
}  /* Print_matrix */
 
 int main() {
  int i,j;

 
  double *A = (double *) malloc( N*M*sizeof(double) );
  double *sal1 = (double *) malloc( N*sizeof(double) );

 
 //rellenar matriz en CPU
  for (j=0;j<N;j++)
    for(i=0;i<M;i++)
   {
      A[i+M*j]=i+j ;
    }

  Print_matrix(A,M,N);

  mediasmatrizcpu(A,sal1);
  
  for (j=0;j<N;j++)
   printf("media columna %d = %f  \n",j,sal1[j]);



double *dev_A, *dev_sal;
double *sal = (double *) malloc( N*sizeof(double) );
hipMalloc((void**) &dev_A, N*M*sizeof(double));
hipMalloc((void**) &dev_sal, N*sizeof(double));

hipMemcpy(dev_A,A,N*M*sizeof(double), hipMemcpyHostToDevice);

mediatriz_gpu<<<1,N>>>(dev_A,dev_sal);

hipMemcpy(sal,dev_sal,N*sizeof(double), hipMemcpyDeviceToHost);


   for (j=0;j<N;j++)
     printf("media columna %d = %f  \n",j,sal[j]);

  free(A);
  free(sal1);
  free(sal);

  hipFree(dev_A),
  hipFree(dev_sal);

  }
