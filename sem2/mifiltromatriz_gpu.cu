
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 8 //columnas
#define M 12 //filas


void Print_matrix(float C[], int n, int m) {
   int i, j;

   for (i = 0; i < n; i++) {
      for (j = 0; j < m; j++)
         printf("%.2e ", C[i+j*n]);
      printf("\n");
   }
}  /* Print_matrix */

void filtrocpu(float *a, float *c)
{
	int i,j;
	for (i=0;i<M-2;i++)
	  for (j=0;j<N-2;j++)
	   c[i+j*(M-2)]=(a[i+(j+1)*M]+a[i+1+(j+1)*M]+a[i+2+(j+1)*M]+a[i+1+(j)*M]+a[i+1+(j+2)*M])/5.0;
	
}

__global__ void filtrogpu(float *a, float *c)
{
   int tidx = threadIdx.x + blockIdx.x * blockDim.x; //indice local
   int tidy= threadIdx.y + blockIdx.y*blockDim.y;
   c[tidx+tidy*(M-2)]=(a[tidx+(tidy+1)*M]+a[tidx+1+(tidy+1)*M]+a[tidx+2+(tidy+1)*M]+a[tidx+1+(tidy)*M]+a[tidx+1+(tidy+2)*M])/5.0;
     
}

 int main() {
 float A[N*M], C[(N-2)*(M-2)], c_host[(N-2)*(M-2)];
 float *dev_a, *dev_c;
 int i,j;

  for (i=0;i<M;i++)
  for (j=0;j<N;j++)
   {
     A[i+j*M]=i+j;
     
    }
printf("A \n");
Print_matrix(A,M,N);
filtrocpu(A,C);

printf("\n C \n");
Print_matrix(C,M-2,N-2);

hipMalloc((void **) &dev_a, M*N*sizeof(float) );
hipMalloc((void **) &dev_c, (M-2)*(N-2)*sizeof(float) );

hipMemcpy(dev_a, A, M*N*sizeof(float), hipMemcpyHostToDevice);

dim3 threadsXblock(M-2,N-2);

filtrogpu<<<1,threadsXblock>>>(dev_a,dev_c);

hipMemcpy(c_host, dev_c,  (M-2)*(N-2)*sizeof(float) , hipMemcpyDeviceToHost);

printf("\n C GPU \n");
Print_matrix(c_host,M-2,N-2);


  }
	
