
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 8
#define BLOCKSIZE 4

void Print_matrix(int C[])
{
  int i, j;

  for (i = 0; i < N; i++)
  {
    for (j = 0; j < N; j++)
      printf("%d ", C[i + j * N]);
    printf("\n");
  }
} /* Print_matrix */

int calcula_max_vector(int *A)
{
  int i, maximo;
  maximo = A[0];
  for (i = 1; i < N; i++)
    if ((A[i] > maximo))
      maximo = A[i];

  return maximo;
}

void calcula_max(int *A, int *sal)
{
  int i, j, maximo;
  maximo = A[0];
  for (i = 0; i < N; i++)
    for (j = 0; j < N; j++)
      if ((A[i + j * N] > maximo))
        maximo = A[i + j * N];

  *sal = maximo;
}

__global__ void kernel(int *A, int *sal)
{
  int i;
  __shared__ int cache[BLOCKSIZE];
  int tid = threadIdx.x;
  int cacheindex = threadIdx.x;
  int col = blockIdx.x;
  int maximo = A[tid + col * N];
  while (tid < N)
  {
    if ((A[tid + col * N] > maximo))
      maximo = A[tid + col * N];
    tid += BLOCKSIZE;
  }
  cache[cacheindex] = maximo;
  __syncthreads();
  i = blockDim.x / 2;
  while (i != 0)
  {
     if (cacheindex < i){
        if ((cache[cacheindex + i] > cache[cacheindex])){
          cache[cacheindex] = cache[cacheindex + i];
        }
      }
     __syncthreads();
     i = i / 2;
  }
  if (cacheindex == 0)
     sal[col] = cache[0];
}

__global__ void kernel2(int *A, int *sal)
{
  int i, tid = blockIdx.x;
  int maximo = A[0 + tid * N];;
  for (i = 0; i < N; i++)
    if ((A[i + tid * N] > maximo))
      maximo = A[i + tid * N];

  sal[tid] = maximo;

}

int main()
{

  int i, j;

  int *A = (int *)malloc(N * N * sizeof(int));
  int salcpu;

  // rellenar matriz de enteros en CPU
  for (i = 0; i < N; i++)
    for (j = 0; j < N; j++)
    {
      A[i + N * j] = rand() % 1000;
    }
  Print_matrix(A);
  calcula_max(A, &salcpu);
  printf(" \n El maximo calculado en cpu es %d ", salcpu);

  // Aqui pon el código para reservar memoria, copiar matriz, llamar kernel, traer resultados,
  //  y lo que sea necesario

  // Comienzo parte GPU

  int *sal = (int *)malloc(N * sizeof(int)); // variable para copiar resultado parcial de gpu a cpu
  // variables para gpu
  int *dev_A;
  int *dev_sal;

  hipMalloc((void **)&dev_A, N * N * sizeof(int));
  hipMalloc((void **)&dev_sal, N * sizeof(int));
  hipMemcpy(dev_A, A, N * N * sizeof(int), hipMemcpyHostToDevice);

  kernel<<<N, BLOCKSIZE>>>(dev_A, dev_sal);

  hipMemcpy(sal, dev_sal, N * sizeof(int), hipMemcpyDeviceToHost);

  int cont = calcula_max_vector(sal);
  printf(" \n El maximo calculado en gpu con cache es %d ", cont);


  ////////////////////////////////////////////////////
  /////////////////////// VERSION 2

  // Comienzo parte GPU

  int *sal2 = (int *)malloc(N * sizeof(int)); // variable para copiar resultado parcial de gpu a cpu
  // variables para gpu
  int *dev_A2;
  int *dev_sal2;

  hipMalloc((void **)&dev_A2, N * N * sizeof(int));
  hipMalloc((void **)&dev_sal2, N * sizeof(int));
  hipMemcpy(dev_A2, A, N * N * sizeof(int), hipMemcpyHostToDevice);

  kernel2<<<N, 1>>>(dev_A2, dev_sal2);

  hipMemcpy(sal2, dev_sal2, N * sizeof(int), hipMemcpyDeviceToHost);

  cont = calcula_max_vector(sal2);
  printf(" \n El maximo calculado en gpu sin cache es %d ", cont);

  free(A);
  free(sal);
  hipFree(dev_A);
  hipFree(dev_sal);
  hipFree(dev_A2);
  hipFree(dev_sal2);
}
