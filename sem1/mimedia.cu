
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 10

////////////////////
/*
 * MI MEDIA
 */
////////////////////
#include <stdio.h>
#define N 10

void media(double *a, double *c)
{
  int i;
  for (i = 0; i < N - 2; i++)
    c[i] = (a[i] + a[i + 1] + a[i + 2]) / 3.0f;
}

__global__ void mediakernel(double *a, double *c)
{
  int tid = blockIdx.x; // indice local coincide con global
  while (tid < N - 2)
  {
    c[tid] = (a[tid] + a[tid + 1] + a[tid + 2]) / 3.0f;
    // tid+=N; // podia hacer un if mas sencillo, o += gridDim.x porque cada hilo hace lo suyo y sale
    tid += gridDim.x;
  }
}

int main()
{
  double a[N], c[N], c_host[N];
  double *dev_a, *dev_c;
  int i;

  // rellenar vectores en CPU
  for (i = 0; i < N; i++)
  {
    a[i] = i * i;
  }

  media(a, c);

  for (i = 0; i < N - 2; i++)
    printf("  %f\n", c[i]);

  hipMalloc((void **)&dev_a, N * sizeof(double));
  hipMalloc((void **)&dev_c, (N - 2) * sizeof(double));

  hipMemcpy(dev_a, a, N * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(dev_c, c_host, (N - 2) * sizeof(double), hipMemcpyHostToDevice);

  mediakernel<<<N - 2, 1>>>(dev_a, dev_c);

  hipMemcpy(c_host, dev_c, (N - 2) * sizeof(double), hipMemcpyDeviceToHost);

  for (i = 0; i < N - 2; i++)
    printf("  %f\n", c_host[i]);

  hipFree(dev_a);
  hipFree(dev_c);
}
