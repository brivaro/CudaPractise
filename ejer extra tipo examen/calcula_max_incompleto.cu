
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 8
#define BLOCKSIZE 4

void Print_matrix(int C[])
{
  int i, j;

  for (i = 0; i < N; i++)
  {
    for (j = 0; j < N; j++)
      printf("%d ", C[i + j * N]);
    printf("\n");
  }
} /* Print_matrix */

int calcula_max_vector(int *A)
{
  int i, j, maximo;
  maximo = A[0];
  for (i = 1; i < N; i++)
    if ((A[i] > maximo))
      maximo = A[i + j * N];

  return maximo;
}

void calcula_max(int *A, int *sal)
{
  int i, j, maximo;
  maximo = A[0];
  for (i = 0; i < N; i++)
    for (j = 0; j < N; j++)
      if ((A[i + j * N] > maximo))
        maximo = A[i + j * N];

  *sal = maximo;
}

__global__ void kernel(int *A, int *sal)
{
  int i, maximo = A[0];
  __shared__ int cache[BLOCKSIZE];
  int tid = threadIdx.x;
  int cacheindex = threadIdx.x;
  int col = blockIdx.x;
  while (tid < N)
  {
    if ((A[tid + col * N] > maximo))
      maximo = A[tid + col * N];
    tid += BLOCKSIZE;
  }
  cache[cacheindex] = maximo;
  __syncthreads();
  i = blockDim.x / 2;
  while (i != 0)
  {
     if (cacheindex < i){
        if ((cache[cacheindex + i] > cache[cacheindex])){
          cache[cacheindex] = cache[cacheindex + i];
        }
      }
     __syncthreads();
     i = i / 2;
  }
  if (cacheindex == 0)
     sal[col] = cache[0];
}


int main()
{

  int i, j;

  int *A = (int *)malloc(N * N * sizeof(int));
  int salcpu;

  // rellenar matriz de enteros en CPU
  for (i = 0; i < N; i++)
    for (j = 0; j < N; j++)
    {
      A[i + N * j] = rand() % 1000;
    }
  Print_matrix(A);
  calcula_max(A, &salcpu);
  printf(" \n El maximo calculado en cpu es %d ", salcpu);

  // Aqui pon el código para reservar memoria, copiar matriz, llamar kernel, traer resultados,
  //  y lo que sea necesario

  // Comienzo parte GPU

  int *sal = (int *)malloc(N * sizeof(int)); // variable para copiar resultado parcial de gpu a cpu
  // variables para gpu
  int *dev_A;
  int *dev_sal;

  hipMalloc((void **)&dev_A, N * N * sizeof(int));
  hipMalloc((void **)&dev_sal, N * sizeof(int));
  hipMemcpy(dev_A, A, N * N * sizeof(int), hipMemcpyHostToDevice);

  kernel<<<N, BLOCKSIZE>>>(dev_A, dev_sal);

  hipMemcpy(sal, dev_sal, N * sizeof(int), hipMemcpyDeviceToHost);

  int cont = calcula_max_vector(sal);
  printf(" \n El maximo calculado en cpu es %d ", cont);

  free(A);
  free(sal);
}
