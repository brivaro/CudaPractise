
#include <hip/hip_runtime.h>
#include <stdio.h>

#define M 5
#define N 6

void mediasmatrizcpu(double *A, double *sal)
{
    int i, j;
    double suma;
    for (j = 0; j < N; j++)
    {
        suma = 0;
        for (i = 0; i < M; i++)
            suma = suma + A[i + j * M];
        sal[j] = suma / double(M);
    }
}

__global__ void kernel(double *A, double *sal)
{
    int i;
    int tid = blockIdx.x;
    double suma;
    suma = 0;
    for (i = 0; i < M; i++)
        suma = suma + A[i + tid * M];
    sal[tid] = suma / double(M);
}

void Print_matrix(double C[], int m, int n)
{
    int i, j;

    for (i = 0; i < m; i++)
    {
        for (j = 0; j < n; j++)
            printf("%.2e ", C[i + j * m]);
        printf("\n");
    }
} /* Print_matrix */

int main()
{
    int i, j;

    double *A = (double *)malloc(N * M * sizeof(double));
    double *sal1 = (double *)malloc(N * sizeof(double));

    // rellenar matriz en CPU
    for (j = 0; j < N; j++)
        for (i = 0; i < M; i++)
        {
            A[i + M * j] = i + j;
        }

    Print_matrix(A, M, N);

    mediasmatrizcpu(A, sal1);

    for (j = 0; j < N; j++)
        printf("media columna %d = %f  \n", j, sal1[j]);



    double *dev_a, *dev_sal11;
    double *sal11 = (double *)malloc(N * sizeof(double));
    hipMalloc((void **)&dev_a, M * N * sizeof(double));
    hipMalloc((void **)&dev_sal11, N * sizeof(double));

    hipMemcpy(dev_a, A, M * N * sizeof(double), hipMemcpyHostToDevice);

    kernel<<<M,1>>>(dev_a, dev_sal11);

    hipMemcpy(sal11, dev_sal11, N * sizeof(double), hipMemcpyDeviceToHost);

    for (j = 0; j < N; j++)
        printf("media columna %d = %f  \n", j, sal11[j]);

    free(A);
    free(sal1);
    free(sal11);
    hipFree(dev_sal11);
    hipFree(dev_a);
}
