

#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 10

__global__ void media(double *A, double *sal)
{
    int tid = blockIdx.x;
    sal[tid] = (A[tid] + A[tid+1] + A[tid+2]) / 3;
}

__global__ void media2(int *A, int *sal)
{
    int tid = blockIdx.x;
    sal[tid] = (A[tid] + A[tid+1] + A[tid+2]) / 3;
}


int main()
{
  double a[N], c[N - 2];
  double *dev_a, *dev_c;
  int i;
  hipMalloc((void **)&dev_a, N * sizeof(double));
  hipMalloc((void **)&dev_c, (N - 2) * sizeof(double));
  // rellenar vectores en CPU
  for (i = 0; i < N; i++)
  {
    a[i] = i * i;
  }
  hipMemcpy(dev_a, a, N * sizeof(double), hipMemcpyHostToDevice);

  media<<<N - 2, 1>>>(dev_a, dev_c);
  hipMemcpy(c, dev_c, (N - 2) * sizeof(double), hipMemcpyDeviceToHost);
  for (i = 0; i < N - 2; i++)
    printf("  %f\n", c[i]);

  int aa[N], cc[N];
  for (i = 0; i < N; i++)
  {
    aa[i] = i;
  }

  int *dev_aa, *dev_cc;
  hipMalloc((void **)&dev_aa, (N) * sizeof(int));
  hipMalloc((void **)&dev_cc, (N - 2) * sizeof(int));
  hipMemcpy(dev_aa, aa, N * sizeof(int), hipMemcpyHostToDevice);
  media2<<<N - 2, 1>>>(dev_aa, dev_cc);
  hipMemcpy(cc, dev_cc, (N - 2) * sizeof(int), hipMemcpyDeviceToHost);
  for (i = 0; i < N - 2; i++)
    printf("  %d\n", cc[i]);

  
}