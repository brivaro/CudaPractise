////////////////////	
/*
* MEDIA DEL PROFE
*/
////////////////////

#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 10

__global__ void media(double *a,  double *c)
{
  int tid=blockIdx.x;
  c[tid]=(a[tid]+a[tid+1]+a[tid+2])/3.0f;
 }
 
 int main() {
 double a[N], c[N-2];
 double *dev_a, *dev_c;
  int i;
hipMalloc((void **) &dev_a, N*sizeof(double ));
 hipMalloc((void **) &dev_c, (N-2)*sizeof(double) );
 //rellenar vectores en CPU
  for (i=0;i<N;i++)
   {
     a[i]=i*i;
    }
hipMemcpy( dev_a, a, N*sizeof(double), hipMemcpyHostToDevice );

 media<<<N-2,1>>>(dev_a,dev_c);
hipMemcpy( c, dev_c, (N-2)*sizeof(double), hipMemcpyDeviceToHost );
 for (i=0;i<N-2;i++)
  printf("  %f\n",  c[i]);
  

  }