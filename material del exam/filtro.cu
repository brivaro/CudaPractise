
#include <hip/hip_runtime.h>
#include <stdio.h>
#define M 8
#define N 12

void Print_matrix(double C[], int n, int m) {
   int i, j;

   for (i = 0; i < n; i++) {
      for (j = 0; j < m; j++)
         printf("%.2e ", C[i+j*n]);
      printf("\n");
   }
}  /* Print_matrix */


__global__ void filtro(double *a, double *b)
{
  int tidx, tidy,tidxp,tidyp;
  tidx=threadIdx.x;
  tidxp=tidx+1;
  tidy=threadIdx.y;
  tidyp=tidy+1;
  if ((tidx <(M-2))&&(tidy<(N-2)))
 {
  b[tidx+tidy*(M-2)]=(a[tidxp+tidyp*M]+a[tidxp-1+tidyp*M]+a[tidxp+1+tidyp*M]+a[tidxp+(tidyp-1)*M]+a[tidxp+(tidyp+1)*M])/5.0f;
  }
 }
 
 int main() {
 double a[N*M], b[(N-2)*(M-2)];
 double *dev_a, *dev_b;
 int i,j;
 //reservar memoria en GPU
 hipMalloc((void **) &dev_a, N*M*sizeof(double) );
 hipMalloc((void **) &dev_b, (N-2)*(M-2)*sizeof(double) );
 
 //rellenar vectores en CPU
  for (i=0;i<M;i++)
  for (j=0;j<N;j++)
   {
     a[i+j*M]=i+j;
     
    }
Print_matrix(a,M,N);
printf("esta era A \n");
//enviar vectores a GPU
hipMemcpy( dev_a, a, N*M*sizeof(double) , hipMemcpyHostToDevice );
//cudaMemcpy( dev_b, b, (N-2)*(M-2)*sizeof(double) , cudaMemcpyHostToDevice );
//cudaMemcpy( dev_c, c, N*sizeof(int) , cudaMemcpyHostToDevice );
//dim3 block_p_grd(2,3);
dim3 thr_p_block((M-2),(N-2));
//llamar al Kernel
 filtro<<<1,thr_p_block>>>(dev_a,dev_b);
 //obtener el resultado de vuelta en la CPU
 hipMemcpy( b, dev_b, (N-2)*(M-2)*sizeof(double), hipMemcpyDeviceToHost );

Print_matrix(b,M-2,N-2);
printf("esta era b \n");

  hipFree(dev_a) ;
  hipFree(dev_b) ;
  
  }
	
	
