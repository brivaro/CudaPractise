
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 10

__global__ void add(int *a, int *b, int *c)
{
  int tid=blockIdx.x; // tid = threadIdx.x + blockIdx.x * blockDim.x 
  if (tid <N) {
  c[tid]=a[tid]+b[tid];
  }
 }
 
 int main() {
 int a[N], b[N], c[N];
 int *dev_a, *dev_b, *dev_c,i;
 //reservar memoria en GPU
 hipMalloc((void **) &dev_a, N*sizeof(int) );
 hipMalloc((void **) &dev_b, N*sizeof(int) );
 hipMalloc((void **) &dev_c, N*sizeof(int) );
 //rellenar vectores en CPU
  for (i=0;i<N;i++)
   {
     a[i]=-i;
     b[i]=i*i;
    }
//enviar vectores a GPU
hipMemcpy( dev_a, a, N*sizeof(int) , hipMemcpyHostToDevice );
hipMemcpy( dev_b, b, N*sizeof(int) , hipMemcpyHostToDevice );
hipMemcpy( dev_c, c, N*sizeof(int) , hipMemcpyHostToDevice );

//llamar al Kernel
 add<<<N,1>>>(dev_a,dev_b,dev_c);
 //obtener el resultado de vuelta en la CPU
 hipMemcpy( c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost );
 for (i=0;i<N;i++)
  printf(" %d + %d = %d\n", a[i],b[i], c[i]);
  
  hipFree(dev_a) ;
  hipFree(dev_b) ;
  hipFree(dev_c) ;
  }
	
	
