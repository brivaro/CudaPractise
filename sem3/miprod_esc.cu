
#include <hip/hip_runtime.h>
#include <stdio.h>
#define CUDA_SAFE_CALL(call)                                                 \
  {                                                                          \
    hipError_t err = call;                                                  \
    if (hipSuccess != err)                                                  \
    {                                                                        \
      fprintf(stderr, "CUDA: error occurred in cuda routine. Exiting...\n"); \
      exit(err);                                                             \
    }                                                                        \
  }
#define BLOCKSIZE 32

__global__ void prod_esc_gpu(float *x, float *y, float *sal, uint N)
{
  __shared__ float cache[BLOCKSIZE];
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int cacheindex = threadIdx.x;

  float suma = 0.0, temp = 0.0;
  while (tid < N)
  {
    temp += x[tid] * y[tid];
    tid += gridDim.x * blockDim.x;
  }
  cache[cacheindex] = temp;

  __syncthreads();

  int i = blockDim.x / 2;
  while (i != 0)
  {
    if (cacheindex < i) // solo lo hacen la mitad de hilos del principio
    {
      cache[cacheindex] += cache[cacheindex + i];
    }
    __syncthreads();
    i = i / 2;
  }

  if (threadIdx.x == 0)
  {
    sal[blockIdx.x] = cache[0];
  }
}

float prodesc_cpu(unsigned int n, float *x, float *y)
{
  int j;
  float suma = 0.0;
  for (j = 0; j < n; j++)
    suma += x[j] * y[j];
  return suma;
}

int main(int argc, char *argv[])
{
  unsigned int n;
  unsigned int j;

  n = 1024000;

  float *x = (float *)malloc(n * sizeof(float));
  float *y = (float *)malloc(n * sizeof(float));

  for (j = 0; j < n; j++)
  {
    x[j] = 2.0f * ((float)rand() / RAND_MAX) - 1.0f;
    y[j] = 2.0f * ((float)rand() / RAND_MAX) - 1.0f;
  }

  hipEvent_t start, stop;
  CUDA_SAFE_CALL(hipEventCreate(&start));
  CUDA_SAFE_CALL(hipEventCreate(&stop));

  printf(" x*y en CPU...\n");
  CUDA_SAFE_CALL(hipEventRecord(start, NULL)); // Record the start event
  float res = prodesc_cpu(n, x, y);
  CUDA_SAFE_CALL(hipEventRecord(stop, NULL)); // Record the stop event
  CUDA_SAFE_CALL(hipEventSynchronize(stop));  // Wait for the stop event to complete
  float msecCPU = 0.0f;
  CUDA_SAFE_CALL(hipEventElapsedTime(&msecCPU, start, stop));

  printf(" x*y en GPU...\n");
  int n_blocks = n / BLOCKSIZE;
  float *dev_x, *dev_y;
  float *dev_sal;
  float *sal = (float *)malloc(n_blocks * sizeof(float));

  hipMalloc((void **)&dev_x, n * sizeof(float));
  hipMalloc((void **)&dev_y, n * sizeof(float));
  hipMalloc((void **)&dev_sal, n_blocks * sizeof(float));

  CUDA_SAFE_CALL(hipMemcpy(dev_x, x, n * sizeof(float), hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(dev_y, y, n * sizeof(float), hipMemcpyHostToDevice));

  // llamada kernel producto escalar

  CUDA_SAFE_CALL(hipEventRecord(start, NULL)); // Record the start event

  prod_esc_gpu<<<n_blocks, BLOCKSIZE>>>(dev_x, dev_y, dev_sal, n);

  CUDA_SAFE_CALL(hipMemcpy(sal, dev_sal, n_blocks * sizeof(float), hipMemcpyDeviceToHost));

  CUDA_SAFE_CALL(hipEventRecord(stop, NULL)); // Record the stop event

  // calculos adicionales ...obtener resultado en variable res_gpu
  float res_gpu = 0.0;
  for (int w = 0; w < n_blocks; w++)
  {
    res_gpu += sal[w];
  }

  CUDA_SAFE_CALL(hipEventSynchronize(stop)); // Wait for the stop event to complete
  float msecGPU = 0.0f;
  CUDA_SAFE_CALL(hipEventElapsedTime(&msecGPU, start, stop));
  printf("CPU time = %.2f msec.\n", msecCPU);
  printf("GPU time = %.2f msec.\n", msecGPU);

  printf("res_cpu %f \n", res);
  printf("res_gpu %f \n", res_gpu);

  free(x);
  free(y);
  hipFree(dev_x);
  hipFree(dev_y);
}
