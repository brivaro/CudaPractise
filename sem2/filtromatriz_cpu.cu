
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 8 //columnas
#define M 12 //filas


void Print_matrix(float C[], int n, int m) {
   int i, j;

   for (i = 0; i < n; i++) {
      for (j = 0; j < m; j++)
         printf("%.2e ", C[i+j*n]);
      printf("\n");
   }
}  /* Print_matrix */

void filtrocpu(float *a, float *c)
{
	int i,j;
	for (i=0;i<M-2;i++)
	  for (j=0;j<N-2;j++)
	   c[i+j*(M-2)]=(a[i+(j+1)*M]+a[i+1+(j+1)*M]+a[i+2+(j+1)*M]+a[i+1+(j)*M]+a[i+1+(j+2)*M])/5.0;
	
}
 
 int main() {
 float A[N*M], C[(N-2)*(M-2)];
 int i,j;

  for (i=0;i<M;i++)
  for (j=0;j<N;j++)
   {
     A[i+j*M]=i+j;
     
    }
printf("A \n");
Print_matrix(A,M,N);
filtrocpu(A,C);

printf("\n C \n");
Print_matrix(C,M-2,N-2);

  }
	
	
