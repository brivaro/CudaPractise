
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 8  // filas
#define M 12 // columnas

__global__ void add(int *a, int *b, int *c)
{
   int tid = threadIdx.x + blockIdx.x * blockDim.x;
   c[tid] = a[tid] + b[tid];
}

void Print_matrix(int C[], int n, int m)
{
   int i, j;

   for (i = 0; i < m; i++)
   {
      for (j = 0; j < n; j++)
         printf("%d ", C[i + j * m]);
      printf("\n");
   }
   printf("\n");
} /* Print_matrix */

int main()
{
   int a[N * M], b[N * M], c[N * M];
   int *dev_a, *dev_b, *dev_c, i, j;
   // reservar memoria en GPU
   hipMalloc((void **)&dev_a, N * M * sizeof(int));
   hipMalloc((void **)&dev_b, N * M * sizeof(int));
   hipMalloc((void **)&dev_c, N * M * sizeof(int));
   // rellenar matriz en CPU
   for (j = 0; j < M; j++)
   {
      for (i = 0; i < N; i++)
      {
         a[i + N * j] = i + j;
         b[i + N * j] = i + j;
      }
   }
   Print_matrix(a, N, M);
   Print_matrix(b, N, M);
   // enviar vectores a GPU
   hipMemcpy(dev_a, a, N * M * sizeof(int), hipMemcpyHostToDevice);
   hipMemcpy(dev_b, b, N * M * sizeof(int), hipMemcpyHostToDevice);
   hipMemcpy(dev_c, c, N * M * sizeof(int), hipMemcpyHostToDevice);

   // llamar al Kernel
   add<<<M, N>>>(dev_a, dev_b, dev_c);
   // obtener el resultado de vuelta en la CPU
   hipMemcpy(c, dev_c, N * M * sizeof(int), hipMemcpyDeviceToHost);
   Print_matrix(c, N, M);

   hipFree(dev_a);
   hipFree(dev_b);
   hipFree(dev_c);
}
