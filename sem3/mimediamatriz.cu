
#include <hip/hip_runtime.h>
#include <stdio.h>

#define CUDA_SAFE_CALL( call ) {                                         \
 hipError_t err = call;                                                 \
 if( hipSuccess != err ) {                                              \
   fprintf(stderr,"CUDA: error occurred in cuda routine. Exiting...\n"); \
   exit(err);                                                            \
 } }

 #define FILAS 10240
 #define COLUMNAS 1024
 #define BLOCKSIZE 32

__global__ void mediasmatriznaive(double *A,  double *sal)
{
  int i,tid=blockIdx.x; 
  double suma=0;
  if (tid <COLUMNAS) {
    for(i=0;i<FILAS;i++)
       suma=suma+A[i+tid*FILAS];
    sal[tid]=suma/double(FILAS);
  }
 }

__global__ void mediasmatrizfast(double *A,  double *sal)
{
  __shared__ double cache[BLOCKSIZE];
  int tid = threadIdx.x; //fila en la que esta indice
  int col = blockIdx.x;
  int cacheindex = threadIdx.x;

  float suma = 0.0, temp = 0.0;
  while (tid < FILAS)
  {
    temp += A[tid+col*FILAS];
    tid += BLOCKSIZE;
  }
  cache[cacheindex] = temp;

  __syncthreads();

  int i = blockDim.x / 2;
  while (i != 0)
  {
    if (cacheindex < i) // solo lo hacen la mitad de hilos del principio
    {
      cache[cacheindex] += cache[cacheindex + i];
    }
    __syncthreads();
    i = i / 2;
  }

  if (threadIdx.x == 0)
  {
    sal[col] = cache[0]/FILAS;
  }
}
 
 int main() {
  int i,j;
 double *dev_A,  *dev_sal;
 
  double *A = (double *) malloc( FILAS * COLUMNAS *sizeof(double) );
  double *sal1 = (double *) malloc( COLUMNAS *sizeof(double) );
double *sal2 = (double *) malloc( COLUMNAS *sizeof(double) );
 //reservar memoria en GPU
 hipMalloc((void **) &dev_A, FILAS * COLUMNAS *sizeof(double) );
 hipMalloc((void **) &dev_sal, COLUMNAS *sizeof(double) );
 
 //rellenar matriz en CPU
  for (j=0;j<COLUMNAS;j++)
    for(i=0;i<FILAS;i++)
   {
      A[i+FILAS*j]=2.0f * ( (double) rand() / RAND_MAX ) ;
    }
//enviar vectores a GPU
hipMemcpy( dev_A, A, FILAS * COLUMNAS *sizeof(double) , hipMemcpyHostToDevice );


//cudaMemcpy( dev_sal, sal, N*sizeof(double) , cudaMemcpyHostToDevice );

//llamar al Kernel1
  hipEvent_t start, stop;
  CUDA_SAFE_CALL( hipEventCreate(&start) );
  CUDA_SAFE_CALL( hipEventCreate(&stop) );
//
CUDA_SAFE_CALL( hipEventRecord(start, NULL) ); // Record the start event

 mediasmatrizfast<<<COLUMNAS,BLOCKSIZE>>>(dev_A,dev_sal);
 //obtener el resultado de vuelta en la CPU
 hipMemcpy( sal1, dev_sal, COLUMNAS *sizeof(double), hipMemcpyDeviceToHost );

CUDA_SAFE_CALL( hipEventRecord(stop, NULL) );  // Record the stop event
  CUDA_SAFE_CALL( hipEventSynchronize(stop) );   // Wait for the stop event to complete
  float msecGPU = 0.0f;
  CUDA_SAFE_CALL( hipEventElapsedTime(&msecGPU, start, stop) );
  printf("GPU time1 = %.2f msec.\n",msecGPU);
//
// kernel 2
//
CUDA_SAFE_CALL( hipEventRecord(start, NULL) ); // Record the start event

 mediasmatriznaive<<<COLUMNAS,1>>>(dev_A,dev_sal);
 //obtener el resultado de vuelta en la CPU
 hipMemcpy( sal2, dev_sal, COLUMNAS*sizeof(double), hipMemcpyDeviceToHost );

  CUDA_SAFE_CALL( hipEventRecord(stop, NULL) );  // Record the stop event
  CUDA_SAFE_CALL( hipEventSynchronize(stop) );   // Wait for the stop event to complete
  msecGPU = 0.0f;
  CUDA_SAFE_CALL( hipEventElapsedTime(&msecGPU, start, stop) );
  printf("GPU time2 = %.2f msec.\n",msecGPU);

//
  double error=fabs(sal1[0]-sal2[0]);
  double aux;
  for (i=1;i<COLUMNAS;i++)
      { aux=fabs(sal1[i]-sal2[i]);
        if (aux>error)
           error=aux;
      }
  printf("error %f  \n",error);
  hipFree(dev_A) ;
  hipFree(dev_sal) ;
  free(A);
  free(sal1);
 free(sal2);
  }
	
	
