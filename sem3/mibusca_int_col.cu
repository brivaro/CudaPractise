
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 16
#define BLOCKSIZE 4

void Print_matrix(int C[], int n)
{
   int i, j;

   for (i = 0; i < n; i++)
   {
      for (j = 0; j < n; j++)
         printf("%d ", C[i + j * n]);
      printf("\n");
   }
} /* Print_matrix */

void contar_int(int *A, int *sal, int num1, int num2)
{
   int i, j, cant = 0;
   for (j = 0; j < N; j++)
      for (i = 0; i < N - 1; i++)
         if ((A[i + j * N] == num1) && (A[i + 1 + j * N] == num2))
            cant++;

   *sal = cant;
}

__global__ void kernel(int *A, int *sal, int num1, int num2)
{
   int cant = 0;
   int i = 0;
   int tid = blockIdx.x;
   for (i = 0; i < N - 1; i++)
      if ((A[i + tid * N] == num1) && (A[i + 1 + tid * N] == num2))
         cant++;

   sal[tid] = cant;
}

__global__ void kernel2(int *A, int *sal, int num1, int num2)
{
   int cant = 0;
   __shared__ int cache[BLOCKSIZE];
   int tid = threadIdx.x;
   int col = blockIdx.x;
   int cacheindex = threadIdx.x;
   while (tid < N)
   {
      if ((A[tid + col * N] == num1) && (A[tid + 1 + col * N] == num2))
         cant++;
      tid += BLOCKSIZE;
   }
   cache[cacheindex] = cant;
   __syncthreads();
   int i = blockDim.x / 2;
   while (i != 0)
   {
      if (cacheindex < i)
         cache[cacheindex] += cache[cacheindex + i];
      __syncthreads();
      i = i / 2;
   }
   if (cacheindex == 0)
      sal[col] = cache[0];
}

int main()
{

   int i, j;

   int *A = (int *)malloc(N * N * sizeof(int));
   int salcpu;

   // rellenar matriz de caracteres en CPU
   for (j = 0; j < N; j++)
      for (i = 0; i < N; i++)
      {
         A[i + N * j] = rand() % 10;
      }
   Print_matrix(A, N);
   contar_int(A, &salcpu, 6, 3);
   printf(" \n En cpu se cuentan %d secuencias %d %d ", salcpu, 6, 3);

   // Aqui pon el código para reservar memoria, copiar matriz, llamar kernel, traer resultados,
   //  y lo que sea necesario

   int *dev_a, *dev_sal;
   int *sal = (int *)malloc(N * sizeof(int)); // variable para copiar resultado de gpu a cpu
   hipMalloc((void **)&dev_a, N * N * sizeof(int));
   hipMalloc((void **)&dev_sal, N * sizeof(int));

   hipMemcpy(dev_a, A, N * N * sizeof(int), hipMemcpyHostToDevice);
   hipMemcpy(dev_sal, sal, N * sizeof(int), hipMemcpyHostToDevice);

   kernel<<<N, 1>>>(dev_a, dev_sal, 6, 3);

   hipMemcpy(sal, dev_sal, N * sizeof(int), hipMemcpyDeviceToHost);

   int res = 0;
   for (i = 0; i < N; i++)
      res += sal[i];

   printf(" \n En gpu se cuentan %d secuencias %d %d ", res, 6, 3);

   ////////////////////////////////////////////
   // version 2
   
   int *dev_A, *dev_SAL;
   int *SAL = (int *)malloc(N * sizeof(int)); // variable para copiar resultado de gpu a cpu
   hipMalloc((void **)&dev_A, N * N * sizeof(int));
   hipMalloc((void **)&dev_SAL, N * sizeof(int));

   hipMemcpy(dev_A, A, N * N * sizeof(int), hipMemcpyHostToDevice);
   hipMemcpy(dev_SAL, SAL, N * sizeof(int), hipMemcpyHostToDevice);

   kernel2<<<N, BLOCKSIZE>>>(dev_A, dev_SAL, 6, 3);

   hipMemcpy(SAL, dev_SAL, N * sizeof(int), hipMemcpyDeviceToHost);

   res = 0;
   for (i = 0; i < N; i++)
      res += SAL[i];

   printf(" \n En gpu se cuentan %d secuencias %d %d ", res, 6, 3);
}
