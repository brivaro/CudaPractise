
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 16
#define BLOCKSIZE 4

void Print_matrix(int C[], int n)
{
   int i, j;

   for (i = 0; i < n; i++)
   {
      for (j = 0; j < n; j++)
         printf("%d ", C[i + j * n]);
      printf("\n");
   }
} /* Print_matrix */

__global__ void contar1(int *A, int *sal, int num1, int num2)
{
   int i, cant = 0;

   int j = blockIdx.x;
   // for (j=0;j<N;j++)
   for (i = 0; i < N - 1; i++)
      if ((A[i + j * N] == num1) && (A[i + 1 + j * N] == num2))
         cant++;

   sal[j] = cant;
}

__global__ void contar2(int *A, int *sal, int num1, int num2)
{
   int i, cant = 0;
   __shared__ double cache[BLOCKSIZE];
   int tid = threadIdx.x;
   int cacheindex = threadIdx.x;
   int col = blockIdx.x;
   while (tid < N - 1)
   {
      if ((A[tid + col * N] == num1) && (A[tid + 1 + col * N] == num2))
         cant++;
      tid += BLOCKSIZE;
   }
   cache[cacheindex] = cant;
   __syncthreads();
   i = blockDim.x / 2;
   while (i != 0)
   {
      if (cacheindex < i)
         cache[cacheindex] += cache[cacheindex + i];
      __syncthreads();
      i = i / 2;
   }
   if (cacheindex == 0)
      sal[col] = cache[0];
}

void contar_int(int *A, int *sal, int num1, int num2)
{
   int i, j, cant = 0;
   for (j = 0; j < N; j++)
      for (i = 0; i < N - 1; i++)
         if ((A[i + j * N] == num1) && (A[i + 1 + j * N] == num2))
            cant++;

   *sal = cant;
}

int main()
{

   int i, j;

   int *A = (int *)malloc(N * N * sizeof(int));
   int salcpu;

   // rellenar matriz de caracteres en CPU
   for (j = 0; j < N; j++)
      for (i = 0; i < N; i++)
      {
         A[i + N * j] = rand() % 10;
      }
   Print_matrix(A, N);
   contar_int(A, &salcpu, 1, 7);
   printf(" \n En cpu se cuentan %d secuencias %d %d ", salcpu, 6, 3);

   // Aqui pon el código para reservar memoria, copiar matriz, llamar kernel, traer resultados,
   //  y lo que sea necesario

   // Comienzo parte GPU

   int *sal = (int *)malloc(N * sizeof(int)); // variable para copiar resultado de gpu a cpu
   // variables para gpu
   int *dev_A;
   int *dev_sal;
   hipMalloc((void **)&dev_A, N * N * sizeof(int));
   hipMalloc((void **)&dev_sal, N * sizeof(int));
   hipMemcpy(dev_A, A, N * N * sizeof(int), hipMemcpyHostToDevice);
   contar1<<<N, 1>>>(dev_A, dev_sal, 1, 7);
   hipMemcpy(sal, dev_sal, N * sizeof(int), hipMemcpyDeviceToHost);
   int cont = 0;
   for (i = 0; i < N; i++)
      cont += sal[i];
   printf(" \n En gpu1 se cuentan %d secuencias %d %d ", cont, 6, 3);

   contar2<<<N, BLOCKSIZE>>>(dev_A, dev_sal, 1, 7);
   hipMemcpy(sal, dev_sal, N * sizeof(int), hipMemcpyDeviceToHost);
   cont = 0;
   for (i = 0; i < N; i++)
      cont += sal[i];
   printf(" \n En gpu2 se cuentan %d secuencias %d %d ", cont, 6, 3);

   free(A);
}
