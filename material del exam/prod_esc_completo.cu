
#include <hip/hip_runtime.h>
#include <stdio.h>
#define CUDA_SAFE_CALL( call ) {                                         \
 hipError_t err = call;                                                 \
 if( hipSuccess != err ) {                                              \
   fprintf(stderr,"CUDA: error occurred in cuda routine. Exiting...\n"); \
   exit(err);                                                            \
 } }
#define	BLOCKSIZE 32


__global__ void producto_esc(float *a, float *b, float *sal, uint n)
{
__shared__ float cache[BLOCKSIZE];
int tid=threadIdx.x+blockIdx.x * blockDim.x;
int cacheindex=threadIdx.x;
float suma=0.0,temp=0.0;
while (tid<n)
{temp +=a[tid]*b[tid];
tid+=gridDim.x * blockDim.x;
}
cache[cacheindex]=temp; 

__syncthreads();
int i=blockDim.x/2;
while (i!=0)
{ if (cacheindex<i)
   cache[cacheindex]+=cache[cacheindex+i];
    __syncthreads();
   i=i/2;
}
if (cacheindex==0)
sal[blockIdx.x]=cache[0];
}

float prodesc_cpu(unsigned int n, float *x, float *y)
{int j;
 float suma=0.0;
   for( j=0; j<n; j++ ) 
     suma+=x[j]*y[j];
  return suma;
}
        


int main( int argc, char *argv[] ) {
  unsigned int n;
  unsigned int j, i;

  n =1024000;

  float *x = (float *) malloc(   n*sizeof(float) );
  float *y = (float *) malloc(   n*sizeof(float) );
  
  for( j=0; j<n; j++ ) {
    x[ j ] = 2.0f * ( (float) rand() / RAND_MAX ) - 1.0f;
    y[ j ] = 2.0f * ( (float) rand() / RAND_MAX ) - 1.0f;
  }

  hipEvent_t start, stop;
  CUDA_SAFE_CALL( hipEventCreate(&start) );
  CUDA_SAFE_CALL( hipEventCreate(&stop) );

  printf(" x*y en CPU...\n");
  CUDA_SAFE_CALL( hipEventRecord(start, NULL) ); // Record the start event
  float res=prodesc_cpu( n, x, y );
  CUDA_SAFE_CALL( hipEventRecord(stop, NULL) );  // Record the stop event
  CUDA_SAFE_CALL( hipEventSynchronize(stop) );   // Wait for the stop event to complete
  float msecCPU = 0.0f;
  CUDA_SAFE_CALL( hipEventElapsedTime(&msecCPU, start, stop) );



  printf(" x*y en GPU...\n");
  int n_blocks = n / BLOCKSIZE ;
  float *dev_x, *dev_y;
  float *dev_sal;
  float *sal= (float *)malloc(n_blocks*sizeof(float) );


 hipMalloc((void **) &dev_x, n*sizeof(float) ) ;
 hipMalloc((void **) &dev_y, n*sizeof(float) ) ;
 hipMalloc((void **) &dev_sal, n_blocks*sizeof(float) ) ;

 
 CUDA_SAFE_CALL( hipMemcpy( dev_x, x,   n*sizeof(float), hipMemcpyHostToDevice ));
 CUDA_SAFE_CALL( hipMemcpy( dev_y, y,   n*sizeof(float), hipMemcpyHostToDevice ));

CUDA_SAFE_CALL( hipEventRecord(start, NULL) ); // Record the start event

  producto_esc<<<n_blocks,BLOCKSIZE>>>(dev_x, dev_y, dev_sal, n);

 CUDA_SAFE_CALL( hipMemcpy( sal, dev_sal, n_blocks *sizeof(float), hipMemcpyDeviceToHost ));

  CUDA_SAFE_CALL( hipEventRecord(stop, NULL) );  // Record the stop event
  
  float res_gpu=0.0;
  for (i=0;i<n_blocks;i++)
    res_gpu+=sal[i];


  CUDA_SAFE_CALL( hipEventSynchronize(stop) );   // Wait for the stop event to complete
  float msecGPU = 0.0f;
  CUDA_SAFE_CALL( hipEventElapsedTime(&msecGPU, start, stop) );
  printf("CPU time = %.2f msec.\n",msecCPU);
  printf("GPU time = %.2f msec.\n",msecGPU);


  printf("res_cpu %f \n",res);
  printf("res_gpu %f \n",res_gpu);
 

  free(x);
  free(y);
  hipFree(dev_x) ;
  hipFree(dev_y) ;
  
  

  
}

