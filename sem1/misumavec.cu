
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 10

__global__ void suma(int *a, int *b, int *c)
{
   int tid = threadIdx.x + blockIdx.x * blockDim.x; //indice local
   while(tid < N){
	c[tid]=a[tid]+b[tid];
	tid += gridDim.x * blockDim.x;
   }

}
int main()
{
	int a[N],b[N], c[N];
    int *dev_a, *dev_b, *dev_c;

	hipMalloc((void**) &dev_a, N*sizeof(int));
	hipMalloc((void**) &dev_b, N*sizeof(int));
	hipMalloc((void**) &dev_c, N*sizeof(int));

	for (int i=0;i<N;i++)
	{
	  a[i]=-i;
	  b[i]=i*i;
	 }

	 for (int i=0;i<N;i++)
       printf(" %d + %d = %d\n", a[i],b[i], a[i]+b[i]);

	 hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice);
	 hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice);

	 suma<<<2,3>>>(dev_a, dev_b, dev_c);

	 hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost);

	 for (int i=0;i<N;i++)
	    printf(" %d + %d = %d\n", a[i],b[i], c[i]);

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	return 0;
}
