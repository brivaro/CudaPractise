
#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void suma(int a, int b, int *c)
{
   *c=a+b;
}
int main()
{
	int c;
	int *dev_c;
	hipMalloc( (void**)&dev_c,sizeof(int) );
	suma<<<1,1>>>(2,7,dev_c);
	hipMemcpy( &c,dev_c, sizeof(int),hipMemcpyDeviceToHost);
	printf("2+7 = %d\n",c);
	hipFree(dev_c);
	return 0;
}
